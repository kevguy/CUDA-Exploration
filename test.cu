#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#define _USE_MATH_DEFINES
#include <iostream>
#include <math.h>
using namespace std;

__device__ __host__ __inline__ float N(float x)
{
  return 0.5 + 0.5 * erf(x*M_SQRT1_2);
}

__device__ __host__ void price(float k, float s, float t, float r, float v, float* c, float* p)
{
  float srt = v* sqrtf(t);
  float d1 = (logf(s/k) + (r+0.5*v*v)*t) / srt;
  float d2 = d1 - srt;
  float kert = k * expf(-r*t);
  *c = N(d1)*s - N(d2)*kert;
  *p = kert - s + *c;
}

__global__ void price(float* k, float* s, float* t, float* r, float* v, float* c, float* p)
{
  int idx = threadIdx.x;
  price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx]);
}

int main()
{
  const int count = 512;
  const int size = count * sizeof(float);
  float *args[5];
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  for (int i = 0; i < 5; ++i)
  {
    hipMalloc(&args[i], size);
    hiprandGenerateUniform(gen, args[i], count);
  }

  float *dc, *dp;
  hipMalloc(&dc, size);
  hipMalloc(&dp, size);

  price<<<1, count>>>(args[0], args[1], args[2], args[3], args[4], dc, dp);

  return 0;
}
